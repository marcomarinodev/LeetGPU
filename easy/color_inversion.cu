#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    const int firstBlockIndex = blockDim.x * blockIdx.x;

    // here it's like accessing a portion of the array, this element 
    // is a pixel, and for each pixel we would like to access 
    // to its RGBA values (reason why we do * 4)
    const int pixelIndex = (threadIdx.x + firstBlockIndex) * 4;

    // number of threads per block can be more then the number of pixels (plus the RGBA values)
    // so we need to check to not to go out of bounds
    if (pixelIndex < width * height * 4) {
        // computing the inverse for only RGB values
        for (int i = 0; i < 3; i++) {
            image[pixelIndex + i] = 255 - image[pixelIndex + i];
        }
    }
}
// image_input, image_output are device pointers (i.e. pointers to memory on the GPU)
void solve(unsigned char* image, int width, int height) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;
    
    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(image, width, height);
    hipDeviceSynchronize();
}